#include "hip/hip_runtime.h"

#include "md5.cu"
#include "sha1.cu"
#include "sha256.cu"
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <stdio.h>
#include <sys/io.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#define cudaCheckErrors(msg)                                                   \
    do {                                                                       \
        hipError_t __err = hipGetLastError();                                \
        if (__err != hipSuccess) {                                            \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,            \
                    hipGetErrorString(__err), __FILE__, __LINE__);            \
            fprintf(stderr, "*** FAILED - ABORTING\n");                        \
            exit(1);                                                           \
        }                                                                      \
    } while (0)

/*-------------
* CONFIGURATION
* ----TYPES---
* -- 1: MD5 --
* -- 2: SHA1 -
* -- 3: SHA256
*-------------*/
#define type 1

#define password_length 30
#if type == 1
#define hash_length 16
#endif
#if type == 2
#define hash_length 20
#endif
#if type == 3
#define hash_length 32
#endif

// Hashesorg
#define num_passwords 446426204

// merged
// #define num_passwords 19922147

char* host_passwords;
int* host_password_lengths;
int* host_start_indexes;

char* device_passwords;
int* device_password_lengths;
int* device_start_indexes;
unsigned char* device_targets;

__device__ int device_num_targets;
__device__ size_t device_password_file_size;

__device__ void print_hash(unsigned char hash[]) {
    int idx;
    for (idx = 0; idx < 32; idx++)
        printf("%02x", hash[idx]);
    printf("\n");
}

__device__ void calculate_hash(unsigned char* pass_cleartext,
                               unsigned char* hash, int length) {

    if (type == 1) {
        MD5_CTX ctx;
        md5_init(&ctx);
        md5_update(&ctx, pass_cleartext, length);
        md5_final(&ctx, hash);
    }

    if (type == 2) {
        SHA1_CTX ctx;
        sha1_init(&ctx);
        sha1_update(&ctx, pass_cleartext, length);
        sha1_final(&ctx, hash);
    }

    if (type == 3) {
        SHA256_CTX ctx;
        sha256_init(&ctx);
        sha256_update(&ctx, pass_cleartext, length);
        sha256_final(&ctx, hash);
    }
}

__global__ void compare_hashes(char* hashes, int* lengths, int* start_indexes,
                               unsigned char* targets) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int numThreads = blockDim.x * gridDim.x;
    int num_to_calculate = num_passwords / numThreads;
    num_to_calculate += 1;

    int i;
    for (i = id; i < num_passwords; i += numThreads) {
        int length = lengths[i];
        int start = start_indexes[i];

        if ((start) < device_password_file_size) {

            unsigned char pass_cleartext[password_length];
            if (length < password_length) {
                memcpy(pass_cleartext, &hashes[start], length);
                pass_cleartext[length] = '\0';

                unsigned char hash[hash_length];
                calculate_hash(pass_cleartext, hash, length);

                for (int k = 0; k < device_num_targets; k++) {
                    bool found = true;
                    for (int j = 0; j < hash_length; j++) {
                        if (hash[j] != targets[k * hash_length + j]) {
                            found = false;
                            break;
                        }
                    }
                    if (found) {
                        printf("Thread %d found it! The password is %s\n", id,
                               pass_cleartext);
                    }
                }
            }
        }
    }
}

int main() {
    FILE* fp;
    if (type == 1) {
        fp = fopen("passwords/eharmony.txt", "r");
    }

    if (type == 2) {
        fp = fopen("passwords/unmasked.lst", "r");
    }

    if (type == 3) {
        fp = fopen("passwords/targets.txt", "r");
    }

    const char* file_name =
        "/datadrive/cracklist/hashesorg/passwords_one_line.txt";
    FILE* length_file =
        fopen("/datadrive/cracklist/hashesorg/password_lengths.txt", "r");

    // Calculate file size
    int fd = open(file_name, O_RDONLY);
    size_t password_file_size;
    password_file_size = lseek(fd, 0, SEEK_END);
    printf("Total file size of the wordlist: %zu \n", password_file_size);

    host_password_lengths = (int*)malloc(num_passwords * sizeof(int));
    host_start_indexes = (int*)malloc(num_passwords * sizeof(int));
    host_passwords =
        (char*)mmap(0, password_file_size, PROT_READ, MAP_PRIVATE, fd, 0);

    // Copy all the lengths into host_password_lengths
    int i = 0;
    int counter = 0;
    int start = 0;
    fscanf(length_file, "%d", &i);
    host_password_lengths[counter] = i;
    host_start_indexes[counter] = 0;
    start += i;
    counter++;
    while (!feof(length_file) && counter < num_passwords) {
        fscanf(length_file, "%d", &i);
        host_password_lengths[counter] = i;
        host_start_indexes[counter] = start;
        start += i;
        counter++;
    }
    fclose(length_file);

    int host_num_targets = 0;
    int ch = 0;
    while (!feof(fp)) {
        ch = fgetc(fp);
        if (ch == '\n') {
            host_num_targets++;
        }
    }

    // host_num_targets = 100000;
    unsigned char* host_targets = (unsigned char*)malloc(
        hash_length * host_num_targets * sizeof(unsigned char));

    char* pos;
    char str[65];

    fseek(fp, 0, SEEK_SET);
    for (int i = 0; i < host_num_targets; i++) {
        if (fgets(str, 100, fp) != NULL) {
            pos = str;
            int count;
            for (count = 0; count < hash_length; count++) {
                sscanf(pos, "%2hhx", &host_targets[i * hash_length + count]);
                pos += 2;
            }
        }
    }

    host_num_targets = 10000;

    hipMemcpyToSymbol(HIP_SYMBOL(device_password_file_size), &password_file_size,
                       sizeof(size_t));
    cudaCheckErrors("After hipMalloc -3");
    hipMemcpyToSymbol(HIP_SYMBOL(device_num_targets), &host_num_targets, sizeof(int));
    cudaCheckErrors("After hipMalloc -2");

    hipMalloc((void**)&device_targets,
               host_num_targets * hash_length * sizeof(unsigned char));
    cudaCheckErrors("After hipMalloc -1");
    hipMemcpy(device_targets, host_targets,
               hash_length * host_num_targets * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    cudaCheckErrors("After hipMemcpy -1");

    hipMalloc((void**)&device_password_lengths, num_passwords * sizeof(int));
    cudaCheckErrors("After hipMalloc 0");
    hipMemcpy(device_password_lengths, host_password_lengths,
               num_passwords * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("After hipMemcpy 0");

    hipMalloc((void**)&device_start_indexes, num_passwords * sizeof(int));
    cudaCheckErrors("After hipMalloc 0.5");
    hipMemcpy(device_start_indexes, host_start_indexes,
               num_passwords * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("After hipMemcpy 0.5");

    hipMalloc((void**)&device_passwords, password_file_size * sizeof(char));
    cudaCheckErrors("After hipMalloc 1");

    hipMemcpy(device_passwords, host_passwords,
               password_file_size * sizeof(char), hipMemcpyHostToDevice);
    cudaCheckErrors("After hipMemcpy 1");

    long numGrid = num_passwords / 1024;
    numGrid += 1;
    dim3 dimGrid(32);
    dim3 dimBlock(1024);
    double n_threads = dimGrid.x * dimBlock.x;
    printf("%ld threads in each grid. Each thread calculating %f hashes \n",
           numGrid, num_passwords / n_threads);

    /* Timing */
    hipEvent_t start_time, stop;
    float elapsedTime;
    hipEventCreate(&start_time);
    hipEventRecord(start_time, 0);

    compare_hashes<<<dimGrid, dimBlock>>>(device_passwords,
                                          device_password_lengths,
                                          device_start_indexes, device_targets);
    cudaCheckErrors("After kernel run ");
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start_time, stop);
    printf("Elapsed time: %f\n", elapsedTime);

    hipFree(device_passwords);
    cudaCheckErrors("After free 1 ");
    hipFree(device_start_indexes);
    cudaCheckErrors("After free 2 ");
    hipFree(device_password_lengths);
    cudaCheckErrors("After free 3 ");

    free(host_start_indexes);
    free(host_password_lengths);

    return EXIT_SUCCESS;
}

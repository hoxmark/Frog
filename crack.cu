#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)

/* Functions to be implemented: */
void device_allocation();
void ftcs_kernel();
void print_gpu_info();

int n_cores = 4992;

void print_gpu_info()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);
    printf("Number of CUDA devices: %d\n", n_devices);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("CUDA device name: %s\n", device_prop.name);
    printf("Compute capability: %d.%d\n", device_prop.major, device_prop.minor);
}

int main(int argc, char **argv)
{
    print_gpu_info();

    {
        int lines_allocated = 999999;
        int max_line_len = 64;

        /* Allocate lines of text */
        char **words = (char **)malloc(sizeof(char *) * lines_allocated);
        if (words == NULL)
        {
            fprintf(stderr, "Out of memory (1).\n");
            exit(1);
        }

        FILE *fp = fopen("crackstation_hash.txt", "r");
        if (fp == NULL)
        {
            fprintf(stderr, "Error opening file.\n");
            exit(2);
        }

        int i;
        for (i = 0; 1; i++)
        {
            int j;

            /* Have we gone over our line allocation? */
            if (i >= lines_allocated)
            {
                printf("We have gone over our line allocation\n"); 
                int new_size;

                /* Double our allocation and re-allocate */
                new_size = lines_allocated * 2;
                words = (char **)realloc(words, sizeof(char *) * new_size);
                if (words == NULL)
                {
                    fprintf(stderr, "Out of memory.\n");
                    exit(3);
                }
                lines_allocated = new_size;
            }
            /* Allocate space for the next line */

            // Sizeof(char)?? 
            words[i] = malloc(max_line_len);
            
            if (words[i] == NULL)
            {
                fprintf(stderr, "Out of memory (3).\n");
                exit(4);
            }
            if (fgets(words[i], max_line_len - 1, fp) == NULL)
                break;

            /* Get rid of CR or LF at end of line */
            for (j = strlen(words[i]) - 1; j >= 0 && (words[i][j] == '\n' || words[i][j] == '\r'); j--)
                ;
            words[i][j + 1] = '\0';
        }
        /* Close file */
        fclose(fp);

        int j;
        for (j = 0; j < i; j++)
            printf("%s\n", words[j]);

        /* Good practice to free memory */
        for (; i >= 0; i--)
            free(words[i]);
        free(words);
        return 0;

        exit(EXIT_SUCCESS);
    }
}